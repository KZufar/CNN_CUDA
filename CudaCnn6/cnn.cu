#include "hip/hip_runtime.h"
#pragma comment (lib, "cublas.lib")
#include "stdio.h"
#include <hip/hip_runtime.h>
using namespace std;
#include <ctime>
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

#include <string>
#include <iomanip>
#include <time.h>
#include <iostream>
#include <cmath>
#include <math.h>

//#include "global.cuh"
#define TRAIN_NUM 60000
#define TEST_NUM 10000
#define ROW 28
#define COL 28
#define CONV_SIZE 24
#define POOL_SIZE 12
#define FC1_SIZE 45
#define FC2_SIZE 10
#define CONV_W_SIZE 5
#define CONV_W_NUM 6

__constant__ float _alpha;
__constant__ int _minibatch;
__constant__ int _epochs;

__device__ int _correct_cnt;
__device__ float _avg_error;

__device__ float _train_image[TRAIN_NUM][ROW][COL];
__device__ int _train_label[TRAIN_NUM];
__device__ float _test_image[TEST_NUM][ROW][COL];
__device__ int _test_label[TEST_NUM];

__device__ float _conv_w[CONV_W_NUM][CONV_W_SIZE][CONV_W_SIZE];
__device__ float _conv_b[CONV_W_NUM];
__device__ float _fc1_b[FC1_SIZE];
__device__ float _fc1_w[FC1_SIZE][CONV_W_NUM][POOL_SIZE][POOL_SIZE];
__device__ float _fc2_b[FC2_SIZE];
__device__ float _fc2_w[FC2_SIZE][FC1_SIZE];

__device__ float _input[ROW][COL];
__device__ float _conv_z[CONV_W_NUM][CONV_SIZE][CONV_SIZE];
__device__ float _conv_a[CONV_W_NUM][CONV_SIZE][CONV_SIZE];
__device__ int _pool_pos[CONV_W_NUM][POOL_SIZE][POOL_SIZE];
__device__ float _pool[CONV_W_NUM][POOL_SIZE][POOL_SIZE];
__device__ float _fc1_z[FC1_SIZE];
__device__ float _fc1_a[FC1_SIZE];
__device__ float _fc2_z[FC2_SIZE];
__device__ float _fc2_a[FC2_SIZE];
__device__ float _output[FC2_SIZE];
__device__ int _answer[FC2_SIZE];

__device__ float _conv_dw[CONV_W_NUM][CONV_W_SIZE][CONV_W_SIZE];
__device__ float _conv_db[CONV_W_NUM];
__device__ float _fc1_db[FC1_SIZE];
__device__ float _fc1_dw[FC1_SIZE][CONV_W_NUM][POOL_SIZE][POOL_SIZE];
__device__ float _fc2_db[FC2_SIZE];
__device__ float _fc2_dw[FC2_SIZE][FC1_SIZE];
__device__ float _C[FC2_SIZE];
__device__ float _fc2_delta[FC2_SIZE];
__device__ float _fc1_delta[FC1_SIZE];
__device__ float _conv_sigma_delta[CONV_W_NUM];
__device__ float _conv_delta[CONV_W_NUM][POOL_SIZE][POOL_SIZE];

__device__ int tmp;


float alpha = 0.2;
int epochs = 5;
int minibatch = 1;

float train_image[TRAIN_NUM][ROW][COL];
int train_label[TRAIN_NUM];
float test_image[TEST_NUM][ROW][COL];
int test_label[TEST_NUM];

float conv_w[CONV_W_NUM][CONV_W_SIZE][CONV_W_SIZE];
float conv_b[CONV_W_NUM];
float fc1_b[FC1_SIZE];
float fc1_w[FC1_SIZE][CONV_W_NUM][POOL_SIZE][POOL_SIZE];
float fc2_b[FC2_SIZE];
float fc2_w[FC2_SIZE][FC1_SIZE];

float input[ROW][COL];
float conv_z[CONV_W_NUM][CONV_SIZE][CONV_SIZE];
float conv_a[CONV_W_NUM][CONV_SIZE][CONV_SIZE];
int pool_pos[CONV_W_NUM][POOL_SIZE][POOL_SIZE];
float pool[CONV_W_NUM][POOL_SIZE][POOL_SIZE];
float fc1_z[FC1_SIZE];
float fc1_a[FC1_SIZE];
float fc2_z[FC2_SIZE];
float fc2_a[FC2_SIZE];
float output[FC2_SIZE];
int answer[FC2_SIZE];

float conv_dw[CONV_W_NUM][CONV_W_SIZE][CONV_W_SIZE];
float conv_db[CONV_W_NUM];
float fc1_db[FC1_SIZE];
float fc1_dw[FC1_SIZE][CONV_W_NUM][POOL_SIZE][POOL_SIZE];
float fc2_db[FC2_SIZE];
float fc2_dw[FC2_SIZE][FC1_SIZE];
float C[FC2_SIZE];
float fc2_delta[FC2_SIZE];
float fc1_delta[FC1_SIZE];
float conv_sigma_delta[CONV_W_NUM];
float conv_delta[CONV_W_NUM][POOL_SIZE][POOL_SIZE];

#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}
int swap_endian(int val)
{
	unsigned char c1, c2, c3, c4;
	c1 = val & 255;
	c2 = (val >> 8) & 255;
	c3 = (val >> 16) & 255;
	c4 = (val >> 24) & 255;
	return ((int)c1 << 24) + ((int)c2 << 16) + ((int)c3 << 8) + c4;
}

float get_rand(float fan_in)
{
	float sum = 0;
	for (int i = 0;i < 12;i++)
		sum += (float)rand() / RAND_MAX;
	sum -= 6;
	sum *= 1 / sqrt(fan_in);
	return sum;
}
void initDevice(int devNum)
{
	int dev = devNum;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));
}

__device__ float _get_rand(int _rand, float fan_in)
{
	float sum = 0;
	for (int i = 0;i < 12;i++)
		sum += (float)_rand / RAND_MAX;
	sum -= 6;
	sum *= 1 / sqrt(fan_in);
	return sum;
}

__device__ float _sigmoid(float x)
{
	return (1 / (1 + exp(-1 * x)));
}

//#include "io.cuh"
void load_data()
{
	FILE* f_images = fopen("D:\\\\Zufar\\\\CUDA-CNN\\\\CudaCNN2\\\\CudaCNN2\\\\data\\\\train-images.idx3-ubyte", "rb");
	FILE* f_labels = fopen("D:\\\\Zufar\\\\CUDA-CNN\\\\CudaCNN2\\\\CudaCNN2\\\\data\\\\train-labels.idx1-ubyte", "rb");

	int tmp;

	int magic_num;
	fread(&magic_num, sizeof(int), 1, f_images);
	fread(&magic_num, sizeof(int), 1, f_labels);

	// printf("debug:%d\n",swap_endian(magic_num));

	int train_size;
	fread(&train_size, sizeof(int), 1, f_images);
	fread(&train_size, sizeof(int), 1, f_labels);
	train_size = swap_endian(train_size);

	// printf("debug:%d\n",swap_endian(train_size));

	int rows, cols;
	fread(&rows, sizeof(int), 1, f_images);
	fread(&cols, sizeof(int), 1, f_images);
	rows = swap_endian(rows);
	cols = swap_endian(cols);

	// printf("debug:%d\n",swap_endian(rows));
	// printf("debug:%d\n",swap_endian(cols));

	for (int i = 0;i < train_size;i++)
	{
		fread(&train_label[i], 1, 1, f_labels);
		if (i % 1000 == 0)
			printf("Training labels : Already read %5d labels\r", i);
		// printf("%d:debug:%d\r",i,train_label[i]);
		// system("pause");
	}
	printf("Training labels : Already read %5d labels\n", train_size);

	for (int i = 0;i < train_size;i++)
	{
		for (int j = 0;j < rows;j++)
			for (int k = 0;k < cols;k++)
			{
				tmp = 0;
				fread(&tmp, 1, 1, f_images);
				train_image[i][j][k] = tmp;
				train_image[i][j][k] /= 255;
				// printf("%d %d %d debug: %f\n",i,j,k,train_image[i][j][k]);
				// system("pause");
			}
		if (i % 1000 == 0)
			printf("Training images : Already read %5d images\r", i);
	}
	printf("Training images : Already read %5d images\n", train_size);

	fclose(f_images);
	fclose(f_labels);

	f_images = fopen("D:\\\\Zufar\\\\CUDA-CNN\\\\CudaCNN2\\\\CudaCNN2\\\\data\\\\t10k-images.idx3-ubyte", "rb");
	f_labels = fopen("D:\\\\Zufar\\\\CUDA-CNN\\\\CudaCNN2\\\\CudaCNN2\\\\data\\\\t10k-labels.idx1-ubyte", "rb");

	fread(&magic_num, sizeof(int), 1, f_images);
	fread(&magic_num, sizeof(int), 1, f_labels);

	int test_size;
	fread(&test_size, sizeof(int), 1, f_images);
	fread(&test_size, sizeof(int), 1, f_labels);
	test_size = swap_endian(test_size);

	fread(&rows, sizeof(int), 1, f_images);
	fread(&cols, sizeof(int), 1, f_images);
	rows = swap_endian(rows);
	cols = swap_endian(cols);

	for (int i = 0;i < test_size;i++)
	{
		fread(&test_label[i], 1, 1, f_labels);
		if (i % 1000 == 0)
			printf("Testing labels : Already read %5d labels\r", i);
	}
	printf("Testing labels : Already read %5d labels\n", test_size);

	for (int i = 0;i < test_size;i++)
	{
		for (int j = 0;j < rows;j++)
			for (int k = 0;k < cols;k++)
			{
				tmp = 0;
				fread(&tmp, 1, 1, f_images);
				test_image[i][j][k] = tmp;
				test_image[i][j][k] /= 255;
			}
		if (i % 1000 == 0)
			printf("Testing images : Already read %5d images\r", i);
	}
	printf("Testing images : Already read %5d images\n\n", test_size);

	fclose(f_images);
	fclose(f_labels);
}

void export_params()
{
	FILE* f_params = fopen("./params.txt", "w");

	fprintf(f_params, "6\n");

	fprintf(f_params, "conv1bias 0 6 ");
	for (int i = 0;i < CONV_W_NUM;i++)
		fprintf(f_params, "%X ", *(int*)& conv_b[i]);
	fprintf(f_params, "\n");

	fprintf(f_params, "conv1filter 0 150 ");
	for (int i = 0;i < CONV_W_NUM;i++)
		for (int j = 0;j < CONV_W_SIZE;j++)
			for (int k = 0;k < CONV_W_SIZE;k++)
				fprintf(f_params, "%X ", *(int*)& conv_w[i][j][k]);
	fprintf(f_params, "\n");

	fprintf(f_params, "ip1bias 0 45 ");
	for (int i = 0;i < FC1_SIZE;i++)
		fprintf(f_params, "%X ", *(int*)& fc1_b[i]);
	fprintf(f_params, "\n");

	fprintf(f_params, "ip1filter 0 38880 ");
	for (int i = 0;i < FC1_SIZE;i++)
		for (int j = 0;j < CONV_W_NUM;j++)
			for (int k = 0;k < POOL_SIZE;k++)
				for (int l = 0;l < POOL_SIZE;l++)
					fprintf(f_params, "%X ", *(int*)& fc1_w[i][j][k][l]);
	fprintf(f_params, "\n");

	fprintf(f_params, "ip2bias 0 10 ");
	for (int i = 0;i < FC2_SIZE;i++)
		fprintf(f_params, "%X ", *(int*)& fc2_b[i]);
	fprintf(f_params, "\n");

	fprintf(f_params, "ip2filter 0 450 ");
	for (int i = 0;i < FC2_SIZE;i++)
		for (int j = 0;j < FC1_SIZE;j++)
			fprintf(f_params, "%X ", *(int*)& fc2_w[i][j]);

	fclose(f_params);

}

//#include "global_gpu.cuh"
//#include "utils_gpu.cuh"
//#include "init_gpu.cuh"

void init_data_gpu()
{
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_train_image), train_image, TRAIN_NUM * ROW * COL * sizeof(float)));
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_train_label), train_label, sizeof(train_label)));
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_test_image), test_image, TEST_NUM * ROW * COL * sizeof(float)));
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_test_label), test_label, sizeof(test_label)));
}

__global__ void init_conv_b(int seed)
{
	int ix = threadIdx.x + blockDim.x * blockIdx.x;
	hiprandState state;
	hiprand_init(seed, ix, 0, &state);
	float rn = _get_rand(abs((int)hiprand(&state)) % RAND_MAX, CONV_W_SIZE * CONV_W_SIZE);
	if (ix < CONV_W_NUM)
		_conv_b[ix] = rn;
}

__global__ void init_conv_w(int seed)
{
	int ix = threadIdx.x + blockDim.x * blockIdx.x;
	int iy = threadIdx.y + blockDim.y * blockIdx.y;
	int iz = threadIdx.z + blockDim.z * blockIdx.z;
	int idx = ix + iy * CONV_W_SIZE + iz * CONV_W_SIZE * CONV_W_SIZE;
	hiprandState state;
	hiprand_init(seed, idx, 0, &state);
	float rn = _get_rand(abs((int)hiprand(&state)) % RAND_MAX, CONV_W_SIZE * CONV_W_SIZE);
	if (ix < CONV_W_NUM && iy < CONV_W_SIZE && iz < CONV_W_SIZE)
		_conv_w[ix][iy][iz] = rn;
}

__global__ void init_fc1_b(int seed)
{
	int ix = threadIdx.x + blockDim.x * blockIdx.x;
	hiprandState state;
	hiprand_init(seed, ix, 0, &state);
	float rn = _get_rand(abs((int)hiprand(&state)) % RAND_MAX, POOL_SIZE * POOL_SIZE * CONV_W_NUM);
	if (ix < FC1_SIZE)
		_fc1_b[ix] = rn;
}

__global__ void init_fc1_w(int seed, int i)
{
	int ix = threadIdx.x + blockDim.x * blockIdx.x;
	int iy = threadIdx.y + blockDim.y * blockIdx.y;
	int iz = threadIdx.z + blockDim.z * blockIdx.z;
	int idx = ix + iy * POOL_SIZE + iz * POOL_SIZE * POOL_SIZE;
	hiprandState state;
	hiprand_init(seed, idx, 0, &state);
	float rn = _get_rand(abs((int)hiprand(&state)) % RAND_MAX, POOL_SIZE * POOL_SIZE * CONV_W_NUM);
	if (ix < CONV_W_NUM && iy < POOL_SIZE && iz < POOL_SIZE)
		_fc1_w[i][ix][iy][iz] = rn;
}

__global__ void init_fc2_b(int seed)
{
	int ix = threadIdx.x + blockDim.x * blockIdx.x;
	hiprandState state;
	hiprand_init(seed, ix, 0, &state);
	float rn = _get_rand(abs((int)hiprand(&state)) % RAND_MAX, FC1_SIZE);
	if (ix < FC2_SIZE)
		_fc2_b[ix] = rn;
}

__global__ void init_fc2_w(int seed)
{
	int ix = threadIdx.x + blockDim.x * blockIdx.x;
	int iy = threadIdx.y + blockDim.y * blockIdx.y;
	int idx = ix + iy * FC1_SIZE;
	hiprandState state;
	hiprand_init(seed, idx, 0, &state);
	float rn = _get_rand(abs((int)hiprand(&state)) % RAND_MAX, FC1_SIZE);
	if (ix < FC2_SIZE && iy < FC1_SIZE)
		_fc2_w[ix][iy] = rn;
}

void init_params_gpu()
{
	srand((unsigned)time(NULL));

	dim3 block1(32);
	dim3 grid1((CONV_W_NUM - 1) / block1.x + 1);
	dim3 block2(32, 32, 32);
	dim3 grid2((CONV_W_NUM - 1) / block2.x + 1, (CONV_W_SIZE - 1) / block2.y + 1, (CONV_W_SIZE - 1) / block2.z + 1);
	dim3 block3(32);
	dim3 grid3((FC1_SIZE - 1) / block3.x + 1);
	dim3 block4(32, 32, 32);
	dim3 grid4((CONV_W_NUM - 1) / block4.x + 1, (POOL_SIZE - 1) / block4.y + 1, (POOL_SIZE - 1) / block4.z + 1);
	dim3 block5(32);
	dim3 grid5((FC2_SIZE - 1) / block5.x + 1);
	dim3 block6(32, 32);
	dim3 grid6((FC2_SIZE - 1) / block6.x + 1, (FC1_SIZE - 1) / block6.y + 1);

	init_conv_b << <block1, grid1 >> > (rand());
	init_conv_w << <block2, grid2 >> > (rand());
	init_fc1_b << <block3, grid3 >> > (rand());

#pragma omp parallel for
	for (int i = 0;i < FC1_SIZE;i++)
		init_fc1_w << <block4, grid4 >> > (rand(), i);
	init_fc2_b << <block5, grid5 >> > (rand());
	init_fc2_w << <block6, grid6 >> > (rand());
	hipDeviceSynchronize();
}
//#include "test_gpu.cuh"
__global__ void test_gpu()
{
	printf("%f %d %d\n", _alpha, _epochs, _minibatch);
	printf("%d\n", tmp);
	tmp = 18;
	printf("%d\n", tmp);
}

__global__ void test_gpu1()
{
	printf("====\n");
	printf("%d\n", tmp);
	tmp = 19;
	printf("%d\n", tmp);
}
//#include "fp_gpu.cuh"

__global__ void _set_input_train(int idx)
{
	int ix = threadIdx.x + blockDim.x * blockIdx.x;
	int iy = threadIdx.y + blockDim.y * blockIdx.y;
	if (ix < ROW && iy < COL)
	{
		_input[ix][iy] = _train_image[idx][ix][iy];
	}
}

__global__ void _set_input_test(int idx)
{
	int ix = threadIdx.x + blockDim.x * blockIdx.x;
	int iy = threadIdx.y + blockDim.y * blockIdx.y;
	if (ix < ROW && iy < COL)
	{
		_input[ix][iy] = _test_image[idx][ix][iy];
	}
}

void set_input_gpu_train(int idx)
{
	dim3 block(32, 32);
	dim3 grid((ROW - 1) / block.x + 1, (COL - 1) / block.y + 1);
	_set_input_train << <block, grid >> > (idx);
	hipDeviceSynchronize();
}

void set_input_gpu_test(int idx)
{
	dim3 block(32, 32);
	dim3 grid((ROW - 1) / block.x + 1, (COL - 1) / block.y + 1);
	_set_input_test << <block, grid >> > (idx);
	hipDeviceSynchronize();
}

__global__ void _input_conv()
{
	int ix = threadIdx.x + blockDim.x * blockIdx.x;
	int iy = threadIdx.y + blockDim.y * blockIdx.y;
	int iz = threadIdx.z + blockDim.z * blockIdx.z;
	if (ix < CONV_W_NUM && iy < CONV_SIZE && iz < CONV_SIZE)
	{
		_conv_z[ix][iy][iz] = 0;
		// #pragma unroll
		for (int l = 0;l < CONV_W_SIZE;l++)
			for (int m = 0;m < CONV_W_SIZE;m++)
				_conv_z[ix][iy][iz] += _input[iy + l][iz + m] * _conv_w[ix][l][m];
		_conv_z[ix][iy][iz] += _conv_b[ix];
		_conv_a[ix][iy][iz] = _sigmoid(_conv_z[ix][iy][iz]);
	}
}

void input_conv_gpu()
{
	dim3 block(8, 8, 8);
	dim3 grid((CONV_W_NUM - 1) / block.x + 1, (CONV_SIZE - 1) / block.y + 1, (CONV_SIZE - 1) / block.z + 1);
	_input_conv << <block, grid >> > ();
	hipDeviceSynchronize();
}

__global__ void _conv_pool()
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int k = threadIdx.z + blockDim.z * blockIdx.z;
	if (i < CONV_W_NUM && j < POOL_SIZE && k < POOL_SIZE)
	{
		float _max = _conv_a[i][j * 2][k * 2];
		_pool_pos[i][j][k] = 0;
		if (_conv_a[i][j * 2][k * 2 + 1] > _max)
		{
			_max = _conv_a[i][j * 2][k * 2 + 1];
			_pool_pos[i][j][k] = 1;
		}
		if (_conv_a[i][j * 2 + 1][k * 2] > _max)
		{
			_max = _conv_a[i][j * 2 + 1][k * 2];
			_pool_pos[i][j][k] = 2;
		}
		if (_conv_a[i][j * 2 + 1][k * 2 + 1] > _max)
		{
			_max = _conv_a[i][j * 2 + 1][k * 2 + 1];
			_pool_pos[i][j][k] = 3;
		}
		_pool[i][j][k] = _max;
	}
}

void conv_pool_gpu()
{
	dim3 block(8, 8, 8);
	dim3 grid((CONV_W_NUM - 1) / block.x + 1, (POOL_SIZE - 1) / block.y + 1, (POOL_SIZE - 1) / block.z + 1);
	_conv_pool << <block, grid >> > ();
	hipDeviceSynchronize();
}

__global__ void _pool_fc1()
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < FC1_SIZE)
	{
		_fc1_z[i] = 0;
		for (int j = 0;j < CONV_W_NUM;j++)
			for (int k = 0;k < POOL_SIZE;k++)
				for (int l = 0;l < POOL_SIZE;l++)
					_fc1_z[i] += _pool[j][k][l] * _fc1_w[i][j][k][l];
		_fc1_z[i] += _fc1_b[i];
		_fc1_a[i] = _sigmoid(_fc1_z[i]);
	}
}

void pool_fc1_gpu()
{
	dim3 block(32);
	dim3 grid((FC1_SIZE - 1) / block.x + 1);
	_pool_fc1 << <block, grid >> > ();
	hipDeviceSynchronize();
}

__global__ void _fc1_fc2()
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < FC2_SIZE)
	{
		_fc2_z[i] = 0;
		for (int j = 0;j < FC1_SIZE;j++)
			_fc2_z[i] += _fc1_a[j] * _fc2_w[i][j];
		_fc2_z[i] += _fc2_b[i];
		_fc2_a[i] = _sigmoid(_fc2_z[i]);
	}
}

void fc1_fc2_gpu()
{
	dim3 block(32);
	dim3 grid((FC2_SIZE - 1) / block.x + 1);
	_fc1_fc2 << <block, grid >> > ();
	hipDeviceSynchronize();
}

__global__ void _set_answer_train(int idx)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < FC2_SIZE)
	{
		_output[i] = _fc2_a[i];
		_answer[i] = (_train_label[idx] == i) ? 1 : 0;
	}
}

__global__ void _set_answer_test(int idx)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < FC2_SIZE)
	{
		_output[i] = _fc2_a[i];
		_answer[i] = (_test_label[idx] == i) ? 1 : 0;
	}
}

void set_answer_gpu_train(int idx)
{
	dim3 block(32);
	dim3 grid((FC2_SIZE - 1) / block.x + 1);
	_set_answer_train << <block, grid >> > (idx);
	hipDeviceSynchronize();
}

void set_answer_gpu_test(int idx)
{
	dim3 block(32);
	dim3 grid((FC2_SIZE - 1) / block.x + 1);
	_set_answer_test << <block, grid >> > (idx);
	hipDeviceSynchronize();
}

__global__ void _check_answer_get_error()
{
	float _max = _output[0];
	int max_pos = 0;
	for (int i = 0;i < FC2_SIZE;i++)
	{
		if (_max < _output[i])
		{
			_max = _output[i];
			max_pos = i;
		}
	}
	if (_answer[max_pos])
		_correct_cnt++;
	for (int i = 0;i < FC2_SIZE;i++)
	{
		_C[i] = _output[i] - _answer[i];
		_avg_error += _C[i] * _C[i] * 0.5;
	}
}

void check_answer_get_error_gpu()
{
	_check_answer_get_error << <1, 1 >> > ();
	hipDeviceSynchronize();
}
//#include "bp_gpu.cuh"

__global__ void _update_fc2_b()
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < FC2_SIZE)
	{
		_fc2_delta[i] = _alpha * _C[i] * (_fc2_a[i] * (1.0 - _fc2_a[i]));
		_fc2_db[i] += _fc2_delta[i];
	}
}

void update_fc2_b_gpu()
{
	dim3 block(32);
	dim3 grid((FC2_SIZE - 1) / block.x + 1);
	_update_fc2_b << <block, grid >> > ();
	hipDeviceSynchronize();
}

__global__ void _update_fc2_w()
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < FC2_SIZE && j < FC1_SIZE)
		_fc2_dw[i][j] += _fc2_delta[i] * _fc1_a[j];
}

void update_fc2_w_gpu()
{
	dim3 block(32, 32);
	dim3 grid((FC2_SIZE - 1) / block.x + 1, (FC1_SIZE - 1) / block.x + 1);
	_update_fc2_w << <block, grid >> > ();
	hipDeviceSynchronize();
}

__global__ void _update_fc1_b()
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < FC1_SIZE)
	{
		float error = 0;
		for (int j = 0;j < FC2_SIZE;j++)
			error += _fc2_delta[j] * _fc2_w[j][i];
		_fc1_delta[i] = error * (_fc1_a[i] * (1.0 - _fc1_a[i]));
		_fc1_db[i] += _fc1_delta[i];
	}
}

void update_fc1_b_gpu()
{
	dim3 block(32);
	dim3 grid((FC1_SIZE - 1) / block.x + 1);
	_update_fc1_b << <block, grid >> > ();
	hipDeviceSynchronize();
}

__global__ void _update_fc1_w(int j)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int k = threadIdx.y + blockDim.y * blockIdx.y;
	int l = threadIdx.z + blockDim.z * blockIdx.z;
	if (i < FC1_SIZE && k < POOL_SIZE && l < POOL_SIZE)
		_fc1_dw[i][j][k][l] += _fc1_delta[i] * _pool[j][k][l];
}

void update_fc1_w_gpu()
{
	dim3 block(8, 8, 8);
	dim3 grid((FC1_SIZE - 1) / block.x + 1, (POOL_SIZE - 1) / block.y + 1, (POOL_SIZE - 1) / block.z + 1);

	// #pragma omp parallel for
	for (int j = 0;j < CONV_W_NUM;j++)
		_update_fc1_w << <block, grid >> > (j);
	hipDeviceSynchronize();
}

__global__ void _update_conv_b()
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < CONV_W_NUM)
	{
		_conv_sigma_delta[i] = 0;
		for (int j = 0;j < POOL_SIZE;j++)
			for (int k = 0;k < POOL_SIZE;k++)
			{
				float error = 0;
				_conv_delta[i][j][k] = 0;
				for (int l = 0;l < FC1_SIZE;l++)
					error += _fc1_delta[l] * _fc1_w[l][i][j][k];
				_conv_delta[i][j][k] = error * (_pool[i][j][k] * (1.0 - _pool[i][j][k]));
				_conv_sigma_delta[i] += error * (_pool[i][j][k] * (1.0 - _pool[i][j][k]));
			}
		_conv_db[i] += _conv_sigma_delta[i];
	}
}

void update_conv_b_gpu()
{
	dim3 block(32);
	dim3 grid((CONV_W_NUM - 1) / block.x + 1);
	_update_conv_b << <block, grid >> > ();
	hipDeviceSynchronize();
}

__global__ void _update_conv_w()
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int k = threadIdx.z + blockDim.z * blockIdx.z;
	if (i < CONV_W_NUM && j < CONV_W_SIZE && k < CONV_W_SIZE)
	{
		float error = 0;
		for (int m = 0;m < POOL_SIZE;m++)
			for (int n = 0;n < POOL_SIZE;n++)
			{
				int x = _pool_pos[i][m][n] / 2;
				int y = _pool_pos[i][m][n] % 2;
				error += _conv_delta[i][m][n] * _input[2 * m + j + x][2 * n + k + y];
			}
		_conv_dw[i][j][k] += error;
	}
}

void update_conv_w_gpu()
{
	dim3 block(8, 8, 8);
	dim3 grid((CONV_W_NUM - 1) / block.x + 1, (CONV_W_SIZE - 1) / block.y + 1, (CONV_W_SIZE - 1) / block.z + 1);
	_update_conv_w << <block, grid >> > ();
	hipDeviceSynchronize();
}

__global__ void assign_fc2_b()
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < FC2_SIZE)
	{
		_fc2_b[i] -= (_fc2_db[i] / _minibatch);
		_fc2_db[i] = 0;
	}
}

__global__ void assign_fc2_w()
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < FC2_SIZE && j < FC1_SIZE)
	{
		_fc2_w[i][j] -= (_fc2_dw[i][j] / _minibatch);
		_fc2_dw[i][j] = 0;
	}
}

__global__ void assign_fc1_b()
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < FC1_SIZE)
	{
		_fc1_b[i] -= (_fc1_db[i] / _minibatch);
		_fc1_db[i] = 0;
	}
}

__global__ void assign_fc1_w(int j)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int k = threadIdx.y + blockDim.y * blockIdx.y;
	int l = threadIdx.z + blockDim.z * blockIdx.z;
	if (i < FC1_SIZE && k < POOL_SIZE && l < POOL_SIZE)
	{
		_fc1_w[i][j][k][l] -= (_fc1_dw[i][j][k][l] / _minibatch);
		_fc1_dw[i][j][k][l] = 0;
	}
}

__global__ void assign_conv_b()
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < CONV_W_NUM)
	{
		_conv_b[i] -= (_conv_db[i] / _minibatch);
		_conv_db[i] = 0;
	}
}

__global__ void assign_conv_w()
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int l = threadIdx.y + blockDim.y * blockIdx.y;
	int m = threadIdx.z + blockDim.z * blockIdx.z;
	if (i < CONV_W_NUM && l < CONV_W_SIZE && m < CONV_W_SIZE)
	{
		_conv_w[i][l][m] -= (_conv_dw[i][l][m] / _minibatch);
		_conv_dw[i][l][m] = 0;
	}
}

void assign_grads_gpu()
{
	dim3 block1(32);
	dim3 grid1((FC2_SIZE - 1) / block1.x + 1);
	assign_fc2_b << <block1, grid1 >> > ();

	dim3 block2(32, 32);
	dim3 grid2((FC2_SIZE - 1) / block2.x + 1, (FC1_SIZE - 1) / block2.y + 1);
	assign_fc2_w << <block2, grid2 >> > ();

	dim3 block3(32);
	dim3 grid3((FC1_SIZE - 1) / block3.x + 1);
	assign_fc1_b << <block3, grid3 >> > ();

	dim3 block4(8, 8, 8);
	dim3 grid4((FC1_SIZE - 1) / block4.x + 1, (POOL_SIZE - 1) / block4.y + 1, (POOL_SIZE - 1) / block4.z + 1);
	for (int j = 0;j < CONV_W_NUM;j++)
		assign_fc1_w << <block4, grid4 >> > (j);

	dim3 block5(32);
	dim3 grid5((CONV_W_NUM - 1) / block5.x + 1);
	assign_conv_b << <block5, grid5 >> > ();

	dim3 block6(8, 8, 8);
	dim3 grid6((CONV_W_NUM - 1) / block6.x + 1, (CONV_W_SIZE - 1) / block6.y + 1, (CONV_W_SIZE - 1) / block6.z + 1);
	assign_conv_w << <block6, grid6 >> > ();

	hipDeviceSynchronize();
}

int correct_cnt;
float avg_error;
float max_acc;

__global__ void _test()
{
	int ix = threadIdx.x + blockDim.x * blockIdx.x;
	int iy = threadIdx.y + blockDim.y * blockIdx.y;
	int iz = threadIdx.z + blockDim.z * blockIdx.z;

	for (int i = 5000;i < 5001;i++)
		for (int j = 0;j < ROW;j++)
		{
			for (int k = 0;k < COL;k++)
				printf("%f ", _test_image[i][j][k]);
			printf("\n");
		}
	printf("%d", _test_label[5000]);

	// printf("%f ",_test_image[ix][iy][iz]);
}

void test()
{
	puts("");
	puts("debug1");
	dim3 block(1, 1, 1);
	dim3 grid(1, 1, 1);
	_test << <block, grid >> > ();
	puts("debug2");
	hipDeviceSynchronize();
	puts("debug3");
}
#define BASE_TYPE int
#define N 1000
#define M 64
__global__ void scalMult(const BASE_TYPE * A, const BASE_TYPE * B, BASE_TYPE * C) {
	BASE_TYPE sum = 0;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	sum = A[idx] * B[idx];
	atomicAdd(C, sum);
}

void scal(int* dev_a, int* dev_b, int* dev_c, dim3 blocksPerGrid) {
	scalMult << <blocksPerGrid, M >> > (dev_a, dev_b, dev_c);
}
int main2(int argc, char* argv[])
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int host_a[N], host_b[N];
	int* host_c = (int*)malloc(sizeof(int));
	int* dev_a, * dev_b, * dev_c, * dev_res;
	cout << "a" << "  " << "b" << endl;
	for (int i = 0; i < N; i++)
	{
		host_a[i] = rand() % 10;
		host_b[i] = rand() % 10;
		//cout << host_a[i] << " " << host_b[i] << endl;
	}
	hipMalloc((void**)& dev_a, N * sizeof(int));
	hipMalloc((void**)& dev_b, N * sizeof(int));
	hipMalloc((void**)& dev_c, sizeof(int));
	hipMemcpy(dev_a, host_a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemset(dev_c, 0, sizeof(int));
	//dim3 threadsPerBlock = dim3(BS, BS);
	dim3 blocksPerGrid = dim3(N / M);
	hipEventRecord(start, 0);
	scal(dev_a, dev_b, dev_c, blocksPerGrid);

	//
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float KernelTime;
	hipEventElapsedTime(&KernelTime, start, stop);
	printf("KernelTme: %.2f millseconds\n", KernelTime);
	hipMemcpy(host_c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("Result: %d", host_c[0]);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("====== aininot260 gh@ysucloud.com ======\n");
	printf("         Processor used : %s\n", argv[1]);
	printf("         Learning rate  : %.2f\n", alpha);
	printf("         Epochs         : %d\n", epochs);
	printf("         Batch size     : %d\n", minibatch);
	printf("========================================\n");
	printf("\n");

	load_data();

	clock_t t = clock();

	//initDevice(0);
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_alpha), &alpha, sizeof(float)));
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_minibatch), &minibatch, sizeof(int)))
		CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_epochs), &epochs, sizeof(int)));
	init_data_gpu();
	set_input_gpu_train(1);
	init_params_gpu();

	for (int i = 1;i <= epochs;i++)
	{

		int value1 = 0;
		float value2 = 0;
		hipMemcpy((void*)& _correct_cnt, &value1, sizeof(int), hipMemcpyHostToDevice);
		CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_correct_cnt), &value1, sizeof(int)));
		hipMemcpy((void*)& _avg_error, &value2, sizeof(int), hipMemcpyHostToDevice);
		CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_avg_error), &value2, sizeof(float)));
		//hipMemcpyToSymbol(HIP_SYMBOL(_correct_cnt), &value1, sizeof(int));
		//hipMemcpyToSymbol(HIP_SYMBOL(_avg_error), &value2, sizeof(float));
		hipDeviceSynchronize();

		for (int j = 0;j < TRAIN_NUM;j++)
		{
			set_input_gpu_train(j);
			input_conv_gpu();
			conv_pool_gpu();
			pool_fc1_gpu();
			fc1_fc2_gpu();
			set_answer_gpu_train(j);
			check_answer_get_error_gpu();

			update_fc2_b_gpu();
			update_fc2_w_gpu();
			update_fc1_b_gpu();
			update_fc1_w_gpu();
			update_conv_b_gpu();
			update_conv_w_gpu();
			if ((j + 1) % minibatch == 0)
				assign_grads_gpu();

			if (j && j % 100 == 0)
			{

				hipMemcpyFromSymbol(&correct_cnt, HIP_SYMBOL(_correct_cnt), sizeof(int));
				hipMemcpyFromSymbol(&avg_error, HIP_SYMBOL(_avg_error), sizeof(float));
				printf("Training  Time spent : %.0fs Image count : %d Accuracy : %0.4f%% Error : %0.4f%% Epoch : %d \r", floor(((float)(clock() - t)) / CLOCKS_PER_SEC), j, ((float)correct_cnt / j) * 100, (avg_error / j) * 100, i);
			}
		}

		hipMemcpyFromSymbol(&correct_cnt, HIP_SYMBOL(_correct_cnt), sizeof(int));
		hipMemcpyFromSymbol(&avg_error, HIP_SYMBOL(_avg_error), sizeof(float));
		printf("Training  Time spent : %.0fs Image count : %d Accuracy : %0.4f%% Error : %0.4f%% Epoch : %d \n", floor(((float)(clock() - t)) / CLOCKS_PER_SEC), TRAIN_NUM, ((float)correct_cnt / TRAIN_NUM) * 100, (avg_error / TRAIN_NUM) * 100, i);

		correct_cnt = 0;
		avg_error = 0;
		hipMemcpyToSymbol(HIP_SYMBOL(_correct_cnt), &correct_cnt, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(_avg_error), &avg_error, sizeof(float));

		for (int j = 0;j < TEST_NUM;j++)
		{
			set_input_gpu_test(j);
			input_conv_gpu();
			conv_pool_gpu();
			pool_fc1_gpu();
			fc1_fc2_gpu();
			set_answer_gpu_test(j);
			check_answer_get_error_gpu();

			if (j && j % 100 == 0)
			{
				hipMemcpyFromSymbol(&correct_cnt, HIP_SYMBOL(_correct_cnt), sizeof(int));
				hipMemcpyFromSymbol(&avg_error, HIP_SYMBOL(_avg_error), sizeof(float));
				printf("Testing   Time spent : %.0fs Image count : %d Accuracy : %0.4f%% Error : %0.4f%% \r", floor(((float)(clock() - t)) / CLOCKS_PER_SEC), j, ((float)correct_cnt / j) * 100, (avg_error / j) * 100);
			}
		}
		hipMemcpyFromSymbol(&correct_cnt, HIP_SYMBOL(_correct_cnt), sizeof(int));
		hipMemcpyFromSymbol(&avg_error, HIP_SYMBOL(_avg_error), sizeof(float));
		printf("Testing   Time spent : %.0fs Image count : %d Accuracy : %0.4f%% Error : %0.4f%% \n", floor(((float)(clock() - t)) / CLOCKS_PER_SEC), TEST_NUM, ((float)correct_cnt / TEST_NUM) * 100, (avg_error / TEST_NUM) * 100);

		if ((float)correct_cnt / TEST_NUM * 100 > max_acc)
		{
			max_acc = (float)correct_cnt / TEST_NUM * 100;
			//export_params();
			printf("The new model has been exported.Accuracy has reached to %0.5f%%\n\n", max_acc);
		}
		else
		{
			alpha = alpha - (alpha / 3);
			hipMemcpyToSymbol(HIP_SYMBOL(_alpha), &alpha, sizeof(float));
			printf("Learning rate has been reduced to %f\n\n", alpha);
		}
	}
	return 0;
}